// SDSC Summer Institute 2020
//
// Andreas Goetz (agoetz@sdsc.edu)

// Hello World Program in CUDA C
//
// Contains a function that is executed on the device (GPU)
// Note that printf statements also work on the GPU
// We are using CUDA variables blockIdx.x and threadIdx.x
// These are unique indices for each thread that is executing on the GPU
// With <<<2,2>>> we launch 2 x 4 = 8 threads (4 threads per block)
//


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void my_kernel(void)
{
  printf("Hello World from GPU! I am thread [%d,%d]\n", blockIdx.x, threadIdx.x);
}

int main(void) {

  my_kernel<<<2,4>>>();
  hipDeviceSynchronize();
  printf("Hello World from CPU!\n");
  return 0;

}
